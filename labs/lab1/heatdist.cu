/*
 *  Please write your name and net ID below
 *  
 *  Last name: Anirudhan
 *  First name: Rajagopalan
 *  Net ID: ajr619
 * 
 */


/* 
 * This file contains the code for doing the heat distribution problem. 
 * You do not need to modify anything except starting  gpu_heat_dist() at the bottom
 * of this file.
 * In gpu_heat_dist() you can organize your data structure and the call to your
 * kernel(s) that you need to write too. 
 * 
 * You compile with:
 * 		nvcc -o heatdist heatdist.cu   
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 

/* To index element (i,j) of a 2D array stored as 1D */
#define index(i, j, N)  ((i)*(N)) + (j)

/* Block width definition */
//define BLOCK_WIDTH 16.0

/*****************************************************************/

// Function declarations: Feel free to add any functions you want.
void  seq_heat_dist(float *, unsigned int, unsigned int);
void  gpu_heat_dist(float *, unsigned int, unsigned int, unsigned int);
float  sum_playground(float *, unsigned int);
float* init_playground(unsigned int);
void do_heat_distribution(unsigned int, unsigned int, unsigned int);

/*****************************************************************/

int main(int argc, char * argv[])
{
	unsigned int N; /* Dimention of NxN matrix */
	unsigned int iterations = 0;
	unsigned int block_sizes[] = {8, 16, 32};

	if(argc != 4)
	{
		fprintf(stderr, "usage: heatdist num  iterations  who\n");
		fprintf(stderr, "num = dimension of the square matrix (50 and up)\n");
		fprintf(stderr, "iterations = number of iterations till stopping (1 and up)\n");
		fprintf(stderr, "who = 0: sequential code on CPU, 1: GPU execution\n");
		exit(1);
	}

	//type_of_device = atoi(argv[3]);
	N = (unsigned int) atoi(argv[1]);
	iterations = (unsigned int) atoi(argv[2]);

	printf("N \t cpu_time \t gpu_time \t cpu_sum \t gpu_sum");

	N = 100;
	while(N < 10 * 1000){
		for(int i = 0; i < 3; i++){
			do_heat_distribution(N, block_sizes[i], iterations);
		}
		N = N * 2;
	}

	return 0;
}

void do_heat_distribution(unsigned int N, unsigned int block_size, unsigned int iterations){
	float* playground;
	float gpu_sum = 0, cpu_sum = 0;

	// to measure time taken by a specific part of the code 
	double cpu_time, gpu_time;
	clock_t start, end;

	playground = init_playground(N);
	//printf("sum is %f \n", sum_playground(playground, N));
	start = clock();
	seq_heat_dist(playground, N, iterations);
	end = clock();

	cpu_time = ((double) (end - start));
	//printf("Time taken for %s is %lf\n", "CPU", cpu_time);

	cpu_sum = sum_playground(playground, N);
	//printf("Sum is %f\n", cpu_sum);

	free(playground);

	playground = init_playground(N);
	//printf("sum is %f \n", sum_playground(playground, N));
	start = clock();
	gpu_heat_dist(playground, N, iterations, block_size); 
	end = clock();    

	gpu_time = ((double) (end - start));
	//printf("Time taken for %s is %lf\n", "GPU", gpu_time);

	gpu_sum = sum_playground(playground, N);
	//printf("Sum is %f\n", gpu_sum);

	free(playground);

	printf("%d \t %lf \t %lf \t %f \t %f \n", N, cpu_time, gpu_time, cpu_sum, gpu_sum);

}

float* init_playground(unsigned int N){
	/* The 2D array of points will be treated as 1D array of NxN elements */
	float * playground; 

	int i;

	/* Dynamically allocate NxN array of floats */
	playground = (float *)calloc(N*N, sizeof(float));
	if( !playground )
	{
		fprintf(stderr, " Cannot allocate the %u x %u array\n", N, N);
		exit(1);
	}

	/* Initialize it: calloc already initalized everything to 0 */
	// Edge elements to 80F
	for(i = 0; i < N; i++){
		playground[index(0,i,N)] = 80;
	}

	for(i = 0; i < N; i++){
		playground[index(i,0,N)] = 80;
	}

	for(i = 0; i < N; i++){
		playground[index(i,N-1, N)] = 80;
	}

	for(i = 0; i < N; i++){
		playground[index(N-1,i,N)] = 80;
	}

	// from (0,10) to (0,30) inclusive are 150F
	for(i = 10; i <= 30 && i < N; i++){
		playground[index(i,0,N)] = 150;
	}
	
	return playground;
}

float  sum_playground(float* playground, unsigned int N){
	int i;
	float sum = 0.0;
	for(i = 0; i < N*N; i++){
		sum = sum + playground[i];
	}
	return sum;
}


/*****************  The CPU sequential version (DO NOT CHANGE THAT) **************/
void  seq_heat_dist(float * playground, unsigned int N, unsigned int iterations)
{
	// Loop indices
	int i, j, k;
	int upper = N-1;

	// number of bytes to be copied between array temp and array playground
	unsigned int num_bytes = 0;

	float * temp; 
	/* Dynamically allocate another array for temp values */
	/* Dynamically allocate NxN array of floats */
	temp = (float *)calloc(N*N, sizeof(float));
	if( !temp )
	{
		fprintf(stderr, " Cannot allocate temp %u x %u array\n", N, N);
		exit(1);
	}

	num_bytes = N*N*sizeof(float);

	/* Copy initial array in temp */
	memcpy((void *)temp, (void *) playground, num_bytes);

	for( k = 0; k < iterations; k++)
	{
		/* Calculate new values and store them in temp */
		for(i = 1; i < upper; i++)
			for(j = 1; j < upper; j++)
				temp[index(i,j,N)] = (playground[index(i-1,j,N)] + 
						playground[index(i+1,j,N)] + 
						playground[index(i,j-1,N)] + 
						playground[index(i,j+1,N)])/4.0;



		/* Move new values into old values */ 
		memcpy((void *)playground, (void *) temp, num_bytes);
	}

}

__global__
void calculate_temperature(float* d_playground, float* d_temp, int N){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if(row < 1 || row >= N-1 || col < 1 || col >= N-1){
		return;
	}

	d_temp[index(row, col, N)] = (d_playground[index(row-1, col, N)] + 
					d_playground[index(row+1, col, N)] + 
					d_playground[index(row, col + 1, N)] + 
					d_playground[index(row, col-1, N)])/4.0;
}

/***************** The GPU version: Write your code here *********************/
__host__
void  gpu_heat_dist(float * playground, unsigned int N, unsigned int iterations, unsigned int BLOCK_WIDTH)
{
	// Loop indices
	//int i, j, k;
	//int upper = N-1;
	int iter = iterations;

	int size = N*N*sizeof(float);
	float *d_playground, *d_temp;

	hipMalloc((void **) &d_playground, size);
	hipMalloc((void **) &d_temp, size );

	hipMemcpy(d_playground, playground, size, hipMemcpyHostToDevice);
	hipMemcpy(d_temp, playground, size, hipMemcpyHostToDevice);

	dim3 dimGrid(ceil(N/(float)BLOCK_WIDTH), ceil(N/(float)BLOCK_WIDTH), 1);
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);

	while(iter > 0){
		calculate_temperature<<<dimGrid, dimBlock>>>(d_playground, d_temp, N);
		hipMemcpy(d_playground, d_temp, size, hipMemcpyDeviceToDevice);
		iter = iter -1;
	}
	hipMemcpy(playground, d_playground, size, hipMemcpyDeviceToHost);

	hipFree(d_temp);
	hipFree(d_playground);
}
