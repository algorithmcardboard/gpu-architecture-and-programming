#include "hip/hip_runtime.h"
/*
 *  Please write your name and net ID below
 *  
 *  Last name: Anirudhan
 *  First name: Rajagopalan
 *  Net ID: ajr619
 * 
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
#include <math.h> 

typedef int TYPE;

TYPE* find_primes(unsigned int);
void do_gpu_seieve(TYPE*, unsigned int);
void fill_zeros(TYPE*, unsigned int);

int main(int argc, char * argv[]){
    unsigned int N;

    if(argc != 2){
        printf("Invoke with just one argument (N) that says the maximum value till which to generate primes.\n");
        return 0;
    }

    N = atoi(argv[1]);

    find_primes(N);
}

__host__ __device__
void fill_zeros(TYPE* arr, unsigned int N){
    unsigned int i = 0;
    for(i= 0; i < N; i++){
        arr[i] = 0;
    }
}

int* findAllPrimes(unsigned int N){
}

__host__
TYPE* find_primes(unsigned int N){
    unsigned int i  = 0;
    TYPE* arr = (TYPE*) malloc(N * sizeof(TYPE));
    fill_zeros(arr);
    int* primes = find_all_primes(sqrt(N));
    // Find all primes till sqrt(n)

    // Call threads with two lists.  One with array of numbers.  The other with all primes till sqrt(n).
}

__global__
void do_gpu_seieve(TYPE* arr, unsigned int N){
}
