/*
 *  Please write your name and net ID below
 *  
 *  Last name: Anirudhan
 *  First name: Rajagopalan
 *  Net ID: ajr619
 * 
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
#include <math.h> 

typedef unsigned int TYPE;

#define BLOCK_WIDTH 1024

TYPE* find_primes(unsigned int, int);
__global__
void do_seieve(TYPE*, unsigned int*, unsigned int, int);
void fill_zeros(TYPE*, unsigned int);

int main(int argc, char * argv[]){
    unsigned int N;
    int i = 0;

    if(argc != 2){
        printf("Invoke with just one argument (N) that says the maximum value till which to generate primes.\n");
        return 0;
    }

    N = atoi(argv[1]);

    //printf("N is %d\n", N);

    unsigned int count = 0;

    TYPE* arr = find_primes(N, 20);
    for(i = 0; i < N; i++){
        if(arr[i] == 0){
            printf("%d ", i + 1);
            count = count + 1;
        }
    }
    printf("Count is %d\n", count);
}

void fill_zeros(TYPE* arr, unsigned int N){
    unsigned int i = 0;
    for(i= 0; i < N; i++){
        arr[i] = 0;
    }
    //printf("Filling zeros");
}

unsigned int* find_next_primes(unsigned int* arr, unsigned int* primes, unsigned int last_prime, int k, unsigned int N){
    int i = 0;
    int j = last_prime + 1;
    for(; j < N && i < k; j++){
        if(arr[j] == 0){
            primes[i++] = j;
        }
    }
    //printf("\n");

    return primes;
}

__host__
TYPE* find_primes(unsigned int N, int k){
    TYPE* arr = (TYPE*) malloc(N * sizeof(TYPE));
    unsigned int* primes = (unsigned int*)malloc(k*sizeof(int));
    unsigned int last_prime = 0;

    fill_zeros(arr, N);
    arr[0] = 1;

	TYPE *d_arr, *d_primes;

	hipMalloc((void **) &d_arr, N*sizeof(TYPE));
	hipMalloc((void **) &d_primes, k*sizeof(int));
    //printf("allocated device memory \n");

	hipMemcpy(d_arr, arr, N*sizeof(TYPE), hipMemcpyHostToDevice);;

    //printf("Copied to device \n");

	dim3 gridDimension(ceil(N/(float)BLOCK_WIDTH), 1, 1);
	dim3 blockDimension(BLOCK_WIDTH, 1, 1);

    int i = 0;

    do{
        find_next_primes(arr, primes, last_prime, k, N);
        for(i = 0; i < k; i++){
            if(*(primes + i) > last_prime){
                last_prime = *(primes + i);
            }
        }
        //printf("primes is %d.  Last prime is %d\n", primes[0], last_prime);
        hipMemcpy(d_primes, primes, k*sizeof(int), hipMemcpyHostToDevice);
        do_seieve<<<gridDimension, blockDimension>>>(d_arr, d_primes, N, k);
        hipMemcpy(arr, d_arr, N*sizeof(TYPE), hipMemcpyDeviceToHost);
    }while(last_prime < (N+1)/2);

    hipMemcpy(arr, d_arr, N*sizeof(TYPE), hipMemcpyDeviceToHost);
    return arr;
}

__global__
void do_seieve(TYPE* d_arr, unsigned int* d_primes, unsigned int N, int k){

    int i, id = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if(id > N){
        return;
    }


    for(i = 0; i < k; i++){
        if(id != (d_primes[i]+1) && (id % (d_primes[i]+1) == 0)){
            d_arr[id -1] = 1;
            //printf("id is %d. prime is %d\n", id, d_primes[i]+1);
        }
    }
}
