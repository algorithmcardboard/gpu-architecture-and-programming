/*
 *  Please write your name and net ID below
 *  
 *  Last name: Anirudhan
 *  First name: Rajagopalan
 *  Net ID: ajr619
 * 
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
#include <math.h> 
#include <hip/hip_runtime_api.h>

typedef unsigned int TYPE;

#define BLOCK_WIDTH 1024
#define PRIME_SIZE 2046

TYPE* find_primes(unsigned int, int);
__global__
void do_seieve(TYPE*, unsigned int*, unsigned int, int);
void fill_zeros(TYPE*, unsigned int);

int main(int argc, char * argv[]){
    unsigned int N;
    int i = 0;

    if(argc != 2){
        printf("Invoke with just one argument (N) that says the maximum value till which to generate primes.\n");
        return 0;
    }

    N = atoi(argv[1]);

    //printf("N is %d\n", N);

    unsigned int count = 0;

    TYPE* arr = find_primes(N, N>10240?PRIME_SIZE:4);

    char buf[12];
    sprintf(buf, "%d.txt", N);
    FILE *fp = fopen(buf,"a");
    if(fp == NULL){
        printf("error opening file");
        return 0;
    }
    for(i = 0; i < N; i++){
        if(arr[i] == 0){
            fprintf(fp, "%d ", i + 1);
            count = count + 1;
        }
    }
    printf("Count is %d\n", count);
}

void fill_zeros(TYPE* arr, unsigned int N){
    unsigned int i = 0;
    for(i= 0; i < N; i++){
        arr[i] = 0;
    }
    //printf("Filling zeros");
}

unsigned int* find_next_primes(unsigned int* arr, unsigned int* primes, unsigned int last_prime, int k, unsigned int N){
    int i = 0;
    int j = last_prime + 1;
    for(; j < N && i < k; j++){
        if(arr[j] == 0){
            primes[i++] = j;
        }
    }
    //printf("\n");

    return primes;
}

__host__
TYPE* find_primes(unsigned int N, int k){
    TYPE* arr = (TYPE*) malloc(N * sizeof(TYPE));
    unsigned int* primes = (unsigned int*)malloc(k*sizeof(int));
    unsigned int last_prime = 0;

    fill_zeros(arr, N);
    arr[0] = 1;

    hipProfilerStart();
	TYPE *d_arr, *d_primes;

	hipMalloc((void **) &d_arr, N*sizeof(TYPE));
	hipMalloc((void **) &d_primes, k*sizeof(int));
    //printf("allocated device memory \n");

	hipMemcpy(d_arr, arr, N*sizeof(TYPE), hipMemcpyHostToDevice);;

    //printf("Copied to device \n");

	dim3 gridDimension(ceil(N/(float)BLOCK_WIDTH), 1, 1);
	dim3 blockDimension(BLOCK_WIDTH, 1, 1);

    int i = 0;

    do{
        find_next_primes(arr, primes, last_prime, k, N);
        for(i = 0; i < k; i++){
            if(*(primes + i) > last_prime){
                last_prime = *(primes + i);
            }
        }
        //printf("primes is %d.  Last prime is %d\n", primes[0], last_prime);
        hipMemcpy(d_primes, primes, k*sizeof(int), hipMemcpyHostToDevice);
        do_seieve<<<gridDimension, blockDimension>>>(d_arr, d_primes, N, k);
        hipMemcpy(arr, d_arr, N*sizeof(TYPE), hipMemcpyDeviceToHost);
    }while(last_prime < (N+1)/2);

    hipMemcpy(arr, d_arr, N*sizeof(TYPE), hipMemcpyDeviceToHost);
    hipProfilerStop();
    return arr;
}

__global__
void do_seieve(TYPE* d_arr, unsigned int* d_primes, unsigned int N, int k){

    int i, id = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if(id > N || d_arr[id-1] == 1 || id < d_primes[0]+1){
        return;
    }


    for(i = 0; i < k; i++){
        if(id != (d_primes[i]+1) && (id % (d_primes[i]+1) == 0)){
            d_arr[id -1] = 1;
            break;
            //printf("id is %d. prime is %d\n", id, d_primes[i]+1);
        }
    }
}
