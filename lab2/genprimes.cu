/*
 *  Please write your name and net ID below
 *  
 *  Last name: Anirudhan
 *  First name: Rajagopalan
 *  Net ID: ajr619
 * 
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
#include <math.h> 

typedef unsigned int TYPE;

#define BLOCK_WIDTH 1024

TYPE* find_primes(unsigned int, int);
__global__
void do_seieve(TYPE*, unsigned int*, unsigned int, int);
void fill_zeros(TYPE*, unsigned int);

int main(int argc, char * argv[]){
    unsigned int N;

    if(argc != 2){
        printf("Invoke with just one argument (N) that says the maximum value till which to generate primes.\n");
        return 0;
    }

    N = atoi(argv[1]);

    find_primes(N, 1);
}

void fill_zeros(TYPE* arr, unsigned int N){
    unsigned int i = 0;
    for(i= 0; i < N; i++){
        arr[i] = 0;
    }
}

unsigned int* find_next_primes(unsigned int* arr, unsigned int* primes, unsigned int last_prime, int k, unsigned int N){
    int i = 0;
    int j = last_prime;
    for(j = last_prime; j < N && i < k; j++){
        if(arr[j] == 0){
            primes[i++] = j + 1;
        }
    }

    return primes;
}

__host__
TYPE* find_primes(unsigned int N, int k){
    TYPE* arr = (TYPE*) malloc(N * sizeof(TYPE));
    fill_zeros(arr, N);

	TYPE *d_arr, *d_primes;

	hipMalloc((void **) &d_arr, N*sizeof(TYPE));
	hipMalloc((void **) &d_primes, k*sizeof(int));

	hipMemcpy(d_arr, arr, N*sizeof(TYPE), hipMemcpyHostToDevice);;

	dim3 gridDimension(ceil(N/(float)BLOCK_WIDTH), 1, 1);
	dim3 blockDimension(BLOCK_WIDTH, 1, 1);

    int i = 0;
    d_arr[0] = 1;

    unsigned int last_prime = 1;
    unsigned int* primes = (unsigned int*)malloc(k*sizeof(int));

    do{
        primes = find_next_primes(arr, primes, last_prime, k, N);
        for(i = 0; i < k; i++){
            if(*(primes + i) > last_prime){
                last_prime = *(primes + i);
            }
        }
        hipMemcpy(d_primes, primes, N*sizeof(int), hipMemcpyHostToDevice);
        do_seieve<<<gridDimension, blockDimension>>>(d_arr, d_primes, N, k);
    }while(last_prime < (N+1)/2);

    hipMemcpy(arr, d_arr, N*sizeof(TYPE), hipMemcpyDeviceToHost);
    return arr;
}

__global__
void do_seieve(TYPE* d_arr, unsigned int* d_primes, unsigned int N, int k){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

    int i = 0;
    for(i = 0; i < k; i++){
        if(id % d_primes[k] == 0){
            d_arr[id] = 1;
        }
    }
}
