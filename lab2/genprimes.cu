#include "hip/hip_runtime.h"
/*
 *  Please write your name and net ID below
 *  
 *  Last name: Anirudhan
 *  First name: Rajagopalan
 *  Net ID: ajr619
 * 
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
#include <math.h> 

typedef unsigned int TYPE;

#define BLOCK_WIDTH 1024;

TYPE* find_primes(unsigned int);
void do_seieve(TYPE*, unsigned int);
void fill_zeros(TYPE*, unsigned int);

int main(int argc, char * argv[]){
    unsigned int N;

    if(argc != 2){
        printf("Invoke with just one argument (N) that says the maximum value till which to generate primes.\n");
        return 0;
    }

    N = atoi(argv[1]);

    find_primes(N);
}

__host__ __device__
void fill_zeros(TYPE* arr, unsigned int N){
    unsigned int i = 0;
    for(i= 0; i < N; i++){
        arr[i] = 0;
    }
}

__host__
TYPE* find_primes(unsigned int N){
    unsigned int i  = 0;
    TYPE* arr = (TYPE*) malloc(N * sizeof(TYPE));
    fill_zeros(arr);

    int* primes = find_all_primes(sqrt(N));

	TYPE *d_arr;

	hipMalloc((void **) &d_arr, N*sizeof(TYPE));

	hipMemcpy(d_arr, arr, N*sizeof(TYPE), hipMemcpyHostToDevice);

	dim3 gridDimension(ceil(N/(float)BLOCK_WIDTH), 1, 1);
	dim3 blockDimension(BLOCK_WIDTH, 1, 1);

    unsigned int nextPrime = 1;
    unsigned int stopValue = (N+1)/2 + 1;

    do{
    }while(nextPrime < (N+1)/2);
}

__global__
void do_seieve(TYPE* arr, unsigned int N){
}
