/*
 *  Please write your name and net ID below
 *  
 *  Last name: Anirudhan
 *  First name: Rajagopalan
 *  Net ID: ajr619
 * 
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 

typedef int TYPE;

void do_gpu_seieve(TYPE*, int);

int main(int argc, char * argv[]){
    unsigned int N;

    if(argc != 2){
        printf("Invoke with just one argument (N) that says the maximum value till which to generate primes.\n");
        return 0;
    }

    N = atoi(argv[1]);
}
